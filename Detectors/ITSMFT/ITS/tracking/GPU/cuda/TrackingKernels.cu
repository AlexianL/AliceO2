#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///

#include <hip/hip_runtime.h>
#include <array>
#include <sstream>
#include <iostream>
#include <unistd.h>
#include <thread>

#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/unique.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>

#include "ITStracking/Constants.h"
#include "ITStracking/Configuration.h"
#include "ITStracking/IndexTableUtils.h"
#include "ITStracking/MathUtils.h"
#include "DataFormatsITS/TrackITS.h"

#include "ITStrackingGPU/TrackerTraitsGPU.h"
#include "ITStrackingGPU/TrackingKernels.h"

#ifndef __HIPCC__
#define THRUST_NAMESPACE thrust::cuda
#else
#define THRUST_NAMESPACE thrust::hip
#endif

#ifdef GPUCA_NO_FAST_MATH
#define GPU_BLOCKS 1
#define GPU_THREADS 1
#else
#define GPU_BLOCKS 99999
#define GPU_THREADS 99999
#endif

// O2 track model
#include "ReconstructionDataFormats/Track.h"
#include "DetectorsBase/Propagator.h"
using namespace o2::track;

#define gpuCheckError(x)                \
  {                                     \
    gpuAssert((x), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
  if (code != hipSuccess) {
    LOGF(error, "GPUassert: %s %s %d", hipGetErrorString(code), file, line);
    if (abort) {
      throw std::runtime_error("GPU assert failed.");
    }
  }
}

namespace o2::its

{
using namespace constants::its2;

namespace gpu
{
GPUd() bool fitTrack(TrackITSExt& track,
                     int start,
                     int end,
                     int step,
                     float chi2clcut,
                     float chi2ndfcut,
                     float maxQoverPt,
                     int nCl,
                     float Bz,
                     const TrackingFrameInfo** tfInfos,
                     const o2::base::Propagator* prop,
                     o2::base::PropagatorF::MatCorrType matCorrType)
{
  for (int iLayer{start}; iLayer != end; iLayer += step) {
    if (track.getClusterIndex(iLayer) == constants::its::UnusedIndex) {
      continue;
    }
    const TrackingFrameInfo& trackingHit = tfInfos[iLayer][track.getClusterIndex(iLayer)];
    if (!track.o2::track::TrackParCovF::rotate(trackingHit.alphaTrackingFrame)) {
      return false;
    }

    if (!prop->propagateToX(track,
                            trackingHit.xTrackingFrame,
                            Bz,
                            o2::base::PropagatorImpl<float>::MAX_SIN_PHI,
                            o2::base::PropagatorImpl<float>::MAX_STEP,
                            matCorrType)) {
      return false;
    }

    if (matCorrType == o2::base::PropagatorF::MatCorrType::USEMatCorrNONE) {
      const float xx0 = (iLayer > 2) ? 1.e-2f : 5.e-3f; // Rough layer thickness
      constexpr float radiationLength = 9.36f;          // Radiation length of Si [cm]
      constexpr float density = 2.33f;                  // Density of Si [g/cm^3]
      if (!track.correctForMaterial(xx0, xx0 * radiationLength * density, true)) {
        return false;
      }
    }

    auto predChi2{track.getPredictedChi2(trackingHit.positionTrackingFrame, trackingHit.covarianceTrackingFrame)};

    if ((nCl >= 3 && predChi2 > chi2clcut) || predChi2 < 0.f) {
      return false;
    }
    track.setChi2(track.getChi2() + predChi2);
    if (!track.o2::track::TrackParCov::update(trackingHit.positionTrackingFrame, trackingHit.covarianceTrackingFrame)) {
      return false;
    }
    nCl++;
  }
  return o2::gpu::GPUCommonMath::Abs(track.getQ2Pt()) < maxQoverPt && track.getChi2() < chi2ndfcut * (nCl * 2 - 5);
}

GPUd() o2::track::TrackParCov buildTrackSeed(const Cluster& cluster1,
                                             const Cluster& cluster2,
                                             const TrackingFrameInfo& tf3,
                                             const float bz)
{
  const float ca = o2::gpu::CAMath::Cos(tf3.alphaTrackingFrame), sa = o2::gpu::CAMath::Sin(tf3.alphaTrackingFrame);
  const float x1 = cluster1.xCoordinate * ca + cluster1.yCoordinate * sa;
  const float y1 = -cluster1.xCoordinate * sa + cluster1.yCoordinate * ca;
  const float z1 = cluster1.zCoordinate;
  const float x2 = cluster2.xCoordinate * ca + cluster2.yCoordinate * sa;
  const float y2 = -cluster2.xCoordinate * sa + cluster2.yCoordinate * ca;
  const float z2 = cluster2.zCoordinate;
  const float x3 = tf3.xTrackingFrame;
  const float y3 = tf3.positionTrackingFrame[0];
  const float z3 = tf3.positionTrackingFrame[1];

  const bool zeroField{o2::gpu::GPUCommonMath::Abs(bz) < o2::constants::math::Almost0};
  const float tgp = zeroField ? o2::gpu::CAMath::ATan2(y3 - y1, x3 - x1) : 1.f;
  const float crv = zeroField ? 1.f : math_utils::computeCurvature(x3, y3, x2, y2, x1, y1);
  const float snp = zeroField ? tgp / o2::gpu::CAMath::Sqrt(1.f + tgp * tgp) : crv * (x3 - math_utils::computeCurvatureCentreX(x3, y3, x2, y2, x1, y1));
  const float tgl12 = math_utils::computeTanDipAngle(x1, y1, x2, y2, z1, z2);
  const float tgl23 = math_utils::computeTanDipAngle(x2, y2, x3, y3, z2, z3);
  const float q2pt = zeroField ? 1.f / o2::track::kMostProbablePt : crv / (bz * o2::constants::math::B2C);
  const float q2pt2 = crv * crv;
  const float sg2q2pt = o2::track::kC1Pt2max * (q2pt2 > 0.0005 ? (q2pt2 < 1 ? q2pt2 : 1) : 0.0005);
  return track::TrackParCov(tf3.xTrackingFrame, tf3.alphaTrackingFrame,
                            {y3, z3, snp, 0.5f * (tgl12 + tgl23), q2pt},
                            {tf3.covarianceTrackingFrame[0],
                             tf3.covarianceTrackingFrame[1], tf3.covarianceTrackingFrame[2],
                             0.f, 0.f, track::kCSnp2max,
                             0.f, 0.f, 0.f, track::kCTgl2max,
                             0.f, 0.f, 0.f, 0.f, sg2q2pt});
}

template <typename T1, typename T2>
struct pair_to_first : public thrust::unary_function<gpuPair<T1, T2>, T1> {
  GPUhd() int operator()(const gpuPair<T1, T2>& a) const
  {
    return a.first;
  }
};

template <typename T1, typename T2>
struct pair_to_second : public thrust::unary_function<gpuPair<T1, T2>, T2> {
  GPUhd() int operator()(const gpuPair<T1, T2>& a) const
  {
    return a.second;
  }
};

template <typename T1, typename T2>
struct is_invalid_pair {
  GPUhd() bool operator()(const gpuPair<T1, T2>& p) const
  {
    return p.first == -1 && p.second == -1;
  }
};

template <typename T1, typename T2>
struct is_valid_pair {
  GPUhd() bool operator()(const gpuPair<T1, T2>& p) const
  {
    return !(p.first == -1 && p.second == -1);
  }
};

template <int nLayers>
GPUg() void fitTrackSeedsKernel(
  CellSeed* trackSeeds,
  const TrackingFrameInfo** foundTrackingFrameInfo,
  o2::its::TrackITSExt* tracks,
  const unsigned int nSeeds,
  const float Bz,
  const int startLevel,
  float maxChi2ClusterAttachment,
  float maxChi2NDF,
  const o2::base::Propagator* propagator,
  const o2::base::PropagatorF::MatCorrType matCorrType)
{
  for (int iCurrentTrackSeedIndex = blockIdx.x * blockDim.x + threadIdx.x; iCurrentTrackSeedIndex < nSeeds; iCurrentTrackSeedIndex += blockDim.x * gridDim.x) {
    auto& seed = trackSeeds[iCurrentTrackSeedIndex];

    TrackITSExt temporaryTrack{seed};

    temporaryTrack.resetCovariance();
    temporaryTrack.setChi2(0);
    int* clusters = seed.getClusters();
    for (int iL{0}; iL < 7; ++iL) {
      temporaryTrack.setExternalClusterIndex(iL, clusters[iL], clusters[iL] != constants::its::UnusedIndex);
    }
    bool fitSuccess = fitTrack(temporaryTrack,               // TrackITSExt& track,
                               0,                            // int lastLayer,
                               nLayers,                      // int firstLayer,
                               1,                            // int firstCluster,
                               maxChi2ClusterAttachment,     // float maxChi2ClusterAttachment,
                               maxChi2NDF,                   // float maxChi2NDF,
                               o2::constants::math::VeryBig, // float maxQoverPt,
                               0,                            // nCl,
                               Bz,                           // float Bz,
                               foundTrackingFrameInfo,       // TrackingFrameInfo** trackingFrameInfo,
                               propagator,                   // const o2::base::Propagator* propagator,
                               matCorrType);                 // o2::base::PropagatorF::MatCorrType matCorrType
    if (!fitSuccess) {
      continue;
    }
    temporaryTrack.getParamOut() = temporaryTrack.getParamIn();
    temporaryTrack.resetCovariance();
    temporaryTrack.setChi2(0);

    fitSuccess = fitTrack(temporaryTrack,           // TrackITSExt& track,
                          nLayers - 1,              // int lastLayer,
                          -1,                       // int firstLayer,
                          -1,                       // int firstCluster,
                          maxChi2ClusterAttachment, // float maxChi2ClusterAttachment,
                          maxChi2NDF,               // float maxChi2NDF,
                          50.f,                     // float maxQoverPt,
                          0,                        // nCl,
                          Bz,                       // float Bz,
                          foundTrackingFrameInfo,   // TrackingFrameInfo** trackingFrameInfo,
                          propagator,               // const o2::base::Propagator* propagator,
                          matCorrType);             // o2::base::PropagatorF::MatCorrType matCorrType
    if (!fitSuccess) {
      continue;
    }
    tracks[iCurrentTrackSeedIndex] = temporaryTrack;
  }
}

template <bool initRun, int nLayers = 7> // Version for new tracker to supersede the old one
GPUg() void computeLayerCellNeighboursKernel(
  CellSeed** cellSeedArray,
  int* neighboursLUT,
  int* neighboursIndexTable,
  int** cellsLUTs,
  gpuPair<int, int>* cellNeighbours,
  const float maxChi2ClusterAttachment,
  const float bz,
  const int layerIndex,
  const unsigned int nCells,
  const int maxCellNeighbours = 1e2)
{
  for (int iCurrentCellIndex = blockIdx.x * blockDim.x + threadIdx.x; iCurrentCellIndex < nCells; iCurrentCellIndex += blockDim.x * gridDim.x) {
    const auto& currentCellSeed{cellSeedArray[layerIndex][iCurrentCellIndex]};
    const int nextLayerTrackletIndex{currentCellSeed.getSecondTrackletIndex()};
    const int nextLayerFirstCellIndex{cellsLUTs[layerIndex + 1][nextLayerTrackletIndex]};
    const int nextLayerLastCellIndex{cellsLUTs[layerIndex + 1][nextLayerTrackletIndex + 1]};
    int foundNeighbours{0};
    for (int iNextCell{nextLayerFirstCellIndex}; iNextCell < nextLayerLastCellIndex; ++iNextCell) {
      CellSeed nextCellSeed{cellSeedArray[layerIndex + 1][iNextCell]};      // Copy
      if (nextCellSeed.getFirstTrackletIndex() != nextLayerTrackletIndex) { // Check if cells share the same tracklet
        break;
      }
      if (!nextCellSeed.rotate(currentCellSeed.getAlpha()) ||
          !nextCellSeed.propagateTo(currentCellSeed.getX(), bz)) {
        continue;
      }
      float chi2 = currentCellSeed.getPredictedChi2(nextCellSeed);
      if (chi2 > maxChi2ClusterAttachment) /// TODO: switch to the chi2 wrt cluster to avoid correlation
      {
        continue;
      }
      if constexpr (initRun) {
        atomicAdd(neighboursLUT + iNextCell, 1);
        foundNeighbours++;
        neighboursIndexTable[iCurrentCellIndex]++;
      } else {
        cellNeighbours[neighboursIndexTable[iCurrentCellIndex] + foundNeighbours] = {iCurrentCellIndex, iNextCell};
        foundNeighbours++;
        // FIXME: this is prone to race conditions: check on level is not atomic
        const int currentCellLevel{currentCellSeed.getLevel()};
        if (currentCellLevel >= nextCellSeed.getLevel()) {
          // atomicExch(cellSeedArray[layerIndex + 1][iNextCell].getLevelPtr(), currentCellLevel + 1); // Update level on corresponding cell
          cellSeedArray[layerIndex + 1][iNextCell].setLevel(currentCellLevel + 1);
        }
      }
    }
  }
}

template <bool initRun, int nLayers = 7>
GPUg() void computeLayerCellsKernel(
  const Cluster** sortedClusters,
  const Cluster** unsortedClusters,
  const TrackingFrameInfo** tfInfo,
  const Tracklet** tracklets,
  const int** trackletsLUT,
  const int nTrackletsCurrent,
  const int layer,
  CellSeed* cells,
  int** cellsLUTs,
  const float bz,
  const float maxChi2ClusterAttachment,
  const float cellDeltaTanLambdaSigma,
  const float nSigmaCut)
{
  constexpr float radl = 9.36f;                                                           // Radiation length of Si [cm].
  constexpr float rho = 2.33f;                                                            // Density of Si [g/cm^3].
  constexpr float layerxX0[7] = {5.e-3f, 5.e-3f, 5.e-3f, 1.e-2f, 1.e-2f, 1.e-2f, 1.e-2f}; // Hardcoded here for the moment.
  for (int iCurrentTrackletIndex = blockIdx.x * blockDim.x + threadIdx.x; iCurrentTrackletIndex < nTrackletsCurrent; iCurrentTrackletIndex += blockDim.x * gridDim.x) {
    const Tracklet& currentTracklet = tracklets[layer][iCurrentTrackletIndex];
    const int nextLayerClusterIndex{currentTracklet.secondClusterIndex};
    const int nextLayerFirstTrackletIndex{trackletsLUT[layer][nextLayerClusterIndex]};
    const int nextLayerLastTrackletIndex{trackletsLUT[layer][nextLayerClusterIndex + 1]};
    if (nextLayerFirstTrackletIndex == nextLayerLastTrackletIndex) {
      continue;
    }
    int foundCells{0};
    for (int iNextTrackletIndex{nextLayerFirstTrackletIndex}; iNextTrackletIndex < nextLayerLastTrackletIndex; ++iNextTrackletIndex) {
      if (tracklets[layer + 1][iNextTrackletIndex].firstClusterIndex != nextLayerClusterIndex) {
        break;
      }
      const Tracklet& nextTracklet = tracklets[layer + 1][iNextTrackletIndex];
      const float deltaTanLambda{o2::gpu::GPUCommonMath::Abs(currentTracklet.tanLambda - nextTracklet.tanLambda)};

      if (deltaTanLambda / cellDeltaTanLambdaSigma < nSigmaCut) {
        const int clusId[3]{
          sortedClusters[layer][currentTracklet.firstClusterIndex].clusterId,
          sortedClusters[layer + 1][nextTracklet.firstClusterIndex].clusterId,
          sortedClusters[layer + 2][nextTracklet.secondClusterIndex].clusterId};

        const auto& cluster1_glo = unsortedClusters[layer][clusId[0]];
        const auto& cluster2_glo = unsortedClusters[layer + 1][clusId[1]];
        const auto& cluster3_tf = tfInfo[layer + 2][clusId[2]];
        auto track{buildTrackSeed(cluster1_glo, cluster2_glo, cluster3_tf, bz)};
        float chi2{0.f};
        bool good{false};
        for (int iC{2}; iC--;) {
          const TrackingFrameInfo& trackingHit = tfInfo[layer + iC][clusId[iC]];
          if (!track.rotate(trackingHit.alphaTrackingFrame)) {
            break;
          }
          if (!track.propagateTo(trackingHit.xTrackingFrame, bz)) {
            break;
          }

          if (!track.correctForMaterial(layerxX0[layer + iC], layerxX0[layer] * radl * rho, true)) {
            break;
          }

          const auto predChi2{track.getPredictedChi2Quiet(trackingHit.positionTrackingFrame, trackingHit.covarianceTrackingFrame)};
          if (!track.o2::track::TrackParCov::update(trackingHit.positionTrackingFrame, trackingHit.covarianceTrackingFrame)) {
            break;
          }
          if (!iC && predChi2 > maxChi2ClusterAttachment) {
            break;
          }
          good = !iC;
          chi2 += predChi2;
        }
        if (!good) {
          continue;
        }
        if constexpr (!initRun) {
          new (cells + cellsLUTs[layer][iCurrentTrackletIndex] + foundCells) CellSeed{layer, clusId[0], clusId[1], clusId[2], iCurrentTrackletIndex, iNextTrackletIndex, track, chi2};
        }
        ++foundCells;
        if constexpr (initRun) {
          cellsLUTs[layer][iCurrentTrackletIndex] = foundCells;
        }
      }
    }
  }
}

/////////////////////////////////////////
// Debug Kernels
/////////////////////////////////////////
GPUd() const int4 getBinsRect(const Cluster& currentCluster, const int layerIndex,
                              const o2::its::IndexTableUtils& utils,
                              const float z1, const float z2, float maxdeltaz, float maxdeltaphi)
{
  const float zRangeMin = o2::gpu::GPUCommonMath::Min(z1, z2) - maxdeltaz;
  const float phiRangeMin = currentCluster.phi - maxdeltaphi;
  const float zRangeMax = o2::gpu::GPUCommonMath::Max(z1, z2) + maxdeltaz;
  const float phiRangeMax = currentCluster.phi + maxdeltaphi;

  if (zRangeMax < -LayersZCoordinate()[layerIndex + 1] ||
      zRangeMin > LayersZCoordinate()[layerIndex + 1] || zRangeMin > zRangeMax) {

    return getEmptyBinsRect();
  }

  return int4{o2::gpu::GPUCommonMath::Max(0, utils.getZBinIndex(layerIndex + 1, zRangeMin)),
              utils.getPhiBinIndex(math_utils::getNormalizedPhi(phiRangeMin)),
              o2::gpu::GPUCommonMath::Min(ZBins - 1, utils.getZBinIndex(layerIndex + 1, zRangeMax)),
              utils.getPhiBinIndex(math_utils::getNormalizedPhi(phiRangeMax))};
}

GPUhd() float Sq(float q)
{
  return q * q;
}

template <typename T>
GPUd() void pPointer(T* ptr)
{
  printf("[%p]\t", ptr);
}
template <typename... Args>
GPUg() void printPointersKernel(std::tuple<Args...> args)
{
  auto print_all = [&](auto... ptrs) {
    (pPointer(ptrs), ...);
  };
  std::apply(print_all, args);
}

// Functors to sort tracklets
template <typename T>
struct trackletSortEmptyFunctor : public thrust::binary_function<T, T, bool> {
  GPUhd() bool operator()(const T& lhs, const T& rhs) const
  {
    return lhs.firstClusterIndex > rhs.firstClusterIndex;
  }
};

template <typename T>
struct trackletSortIndexFunctor : public thrust::binary_function<T, T, bool> {
  GPUhd() bool operator()(const T& lhs, const T& rhs) const
  {
    return lhs.firstClusterIndex < rhs.firstClusterIndex || (lhs.firstClusterIndex == rhs.firstClusterIndex && lhs.secondClusterIndex < rhs.secondClusterIndex);
  }
};

// Print layer buffer
GPUg() void printBufferLayerOnThread(const int layer, const int* v, unsigned int size, const int len = 150, const unsigned int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    for (int i{0}; i < size; ++i) {
      if (!(i % len)) {
        printf("\n layer %d: ===> %d/%d\t", layer, i, (int)size);
      }
      printf("%d\t", v[i]);
    }
    printf("\n");
  }
}

GPUg() void printMatrixRow(const int row, int** mat, const unsigned int rowLength, const int len = 150, const unsigned int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    for (int i{0}; i < rowLength; ++i) {
      if (!(i % len)) {
        printf("\n row %d: ===> %d/%d\t", row, i, (int)rowLength);
      }
      printf("%d\t", mat[row][i]);
    }
    printf("\n");
  }
}

GPUg() void printBufferPointersLayerOnThread(const int layer, void** v, unsigned int size, const int len = 150, const unsigned int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    for (int i{0}; i < size; ++i) {
      if (!(i % len)) {
        printf("\n layer %d: ===> %d/%d\t", layer, i, (int)size);
      }
      printf("%p\t", (void*)v[i]);
    }
    printf("\n");
  }
}

// Dump vertices
GPUg() void printVertices(const Vertex* v, unsigned int size, const unsigned int tId = 0)
{
  if (blockIdx.x * blockDim.x + threadIdx.x == tId) {
    printf("vertices: ");
    for (int i{0}; i < size; ++i) {
      printf("x=%f y=%f z=%f\n", v[i].getX(), v[i].getY(), v[i].getZ());
    }
  }
}

// Dump tracklets
GPUg() void printTracklets(const Tracklet* t,
                           const int offset,
                           const int startRof,
                           const int nrof,
                           const int* roFrameClustersCurrentLayer, // Number of clusters on layer 0 per ROF
                           const int* roFrameClustersNextLayer,    // Number of clusters on layer 1 per ROF
                           const int maxClustersPerRof = 5e2,
                           const int maxTrackletsPerCluster = 50,
                           const unsigned int tId = 0)
{
  if (threadIdx.x == tId) {
    auto offsetCurrent{roFrameClustersCurrentLayer[offset]};
    auto offsetNext{roFrameClustersNextLayer[offset]};
    auto offsetChunk{(startRof - offset) * maxClustersPerRof * maxTrackletsPerCluster};
    for (int i{offsetChunk}; i < offsetChunk + nrof * maxClustersPerRof * maxTrackletsPerCluster; ++i) {
      if (t[i].firstClusterIndex != -1) {
        t[i].dump(offsetCurrent, offsetNext);
      }
    }
  }
}

GPUg() void printTrackletsNotStrided(const Tracklet* t,
                                     const int offset,
                                     const int* roFrameClustersCurrentLayer, // Number of clusters on layer 0 per ROF
                                     const int* roFrameClustersNextLayer,    // Number of clusters on layer 1 per ROF
                                     const int ntracklets,
                                     const unsigned int tId = 0)
{
  if (threadIdx.x == tId) {
    auto offsetCurrent{roFrameClustersCurrentLayer[offset]};
    auto offsetNext{roFrameClustersNextLayer[offset]};
    for (int i{0}; i < ntracklets; ++i) {
      t[i].dump(offsetCurrent, offsetNext);
    }
  }
}

GPUg() void printNeighbours(const gpuPair<int, int>* neighbours,
                            const int* nNeighboursIndexTable,
                            const unsigned int nCells,
                            const unsigned int tId = 0)
{
  for (unsigned int iNeighbour{0}; iNeighbour < nNeighboursIndexTable[nCells]; ++iNeighbour) {
    if (threadIdx.x == tId) {
      printf("%d -> %d\n", neighbours[iNeighbour].first, neighbours[iNeighbour].second);
    }
  }
}

// Compute the tracklets for a given layer
template <int nLayers = 7>
GPUg() void computeLayerTrackletsKernelSingleRof(
  const short rof0,
  const short maxRofs,
  const int layerIndex,
  const Cluster* clustersCurrentLayer,        // input data rof0
  const Cluster* clustersNextLayer,           // input data rof0-delta <rof0< rof0+delta (up to 3 rofs)
  const int* indexTable,                      // input data rof0-delta <rof0< rof0+delta (up to 3 rofs)
  const int* roFrameClusters,                 // input data O(1)
  const int* roFrameClustersNext,             // input data O(1)
  const unsigned char* usedClustersLayer,     // input data rof0
  const unsigned char* usedClustersNextLayer, // input data rof1
  const Vertex* vertices,                     // input data
  int* trackletsLookUpTable,                  // output data
  Tracklet* tracklets,                        // output data
  const int nVertices,
  const int currentLayerClustersSize,
  const float phiCut,
  const float minR,
  const float maxR,
  const float meanDeltaR,
  const float positionResolution,
  const float mSAngle,
  const StaticTrackingParameters<nLayers>* trkPars,
  const IndexTableUtils* utils,
  const unsigned int maxTrackletsPerCluster = 50)
{
  for (int currentClusterIndex = blockIdx.x * blockDim.x + threadIdx.x; currentClusterIndex < currentLayerClustersSize; currentClusterIndex += blockDim.x * gridDim.x) {
    unsigned int storedTracklets{0};
    const Cluster& currentCluster{clustersCurrentLayer[currentClusterIndex]};
    const int currentSortedIndex{roFrameClusters[rof0] + currentClusterIndex};
    if (usedClustersLayer[currentSortedIndex]) {
      continue;
    }
    short minRof = (rof0 >= trkPars->DeltaROF) ? rof0 - trkPars->DeltaROF : 0;
    short maxRof = (rof0 == static_cast<short>(maxRofs - trkPars->DeltaROF)) ? rof0 : rof0 + trkPars->DeltaROF;
    const float inverseR0{1.f / currentCluster.radius};
    for (int iPrimaryVertex{0}; iPrimaryVertex < nVertices; iPrimaryVertex++) {
      const auto& primaryVertex{vertices[iPrimaryVertex]};
      if (primaryVertex.getX() == 0.f && primaryVertex.getY() == 0.f && primaryVertex.getZ() == 0.f) {
        continue;
      }
      const float resolution{o2::gpu::GPUCommonMath::Sqrt(Sq(trkPars->PVres) / primaryVertex.getNContributors() + Sq(positionResolution))};
      const float tanLambda{(currentCluster.zCoordinate - primaryVertex.getZ()) * inverseR0};
      const float zAtRmin{tanLambda * (minR - currentCluster.radius) + currentCluster.zCoordinate};
      const float zAtRmax{tanLambda * (maxR - currentCluster.radius) + currentCluster.zCoordinate};
      const float sqInverseDeltaZ0{1.f / (Sq(currentCluster.zCoordinate - primaryVertex.getZ()) + 2.e-8f)}; /// protecting from overflows adding the detector resolution
      const float sigmaZ{o2::gpu::CAMath::Sqrt(Sq(resolution) * Sq(tanLambda) * ((Sq(inverseR0) + sqInverseDeltaZ0) * Sq(meanDeltaR) + 1.f) + Sq(meanDeltaR * mSAngle))};

      const int4 selectedBinsRect{getBinsRect(currentCluster, layerIndex, *utils, zAtRmin, zAtRmax, sigmaZ * trkPars->NSigmaCut, phiCut)};
      if (selectedBinsRect.x == 0 && selectedBinsRect.y == 0 && selectedBinsRect.z == 0 && selectedBinsRect.w == 0) {
        continue;
      }
      int phiBinsNum{selectedBinsRect.w - selectedBinsRect.y + 1};
      if (phiBinsNum < 0) {
        phiBinsNum += trkPars->PhiBins;
      }
      constexpr int tableSize{256 * 128 + 1}; // hardcoded for the time being

      for (short rof1{minRof}; rof1 <= maxRof; ++rof1) {
        if (!(roFrameClustersNext[rof1 + 1] - roFrameClustersNext[rof1])) { // number of clusters on next layer > 0
          continue;
        }
        for (int iPhiCount{0}; iPhiCount < phiBinsNum; iPhiCount++) {
          int iPhiBin = (selectedBinsRect.y + iPhiCount) % trkPars->PhiBins;
          const int firstBinIndex{utils->getBinIndex(selectedBinsRect.x, iPhiBin)};
          const int maxBinIndex{firstBinIndex + selectedBinsRect.z - selectedBinsRect.x + 1};
          const int firstRowClusterIndex = indexTable[rof1 * tableSize + firstBinIndex];
          const int maxRowClusterIndex = indexTable[rof1 * tableSize + maxBinIndex];
          for (int iNextCluster{firstRowClusterIndex}; iNextCluster < maxRowClusterIndex; ++iNextCluster) {
            if (iNextCluster >= (roFrameClustersNext[rof1 + 1] - roFrameClustersNext[rof1])) {
              break;
            }
            const Cluster& nextCluster{getPtrFromRuler<Cluster>(rof1, clustersNextLayer, roFrameClustersNext)[iNextCluster]};
            if (usedClustersNextLayer[nextCluster.clusterId]) {
              continue;
            }
            const float deltaPhi{o2::gpu::GPUCommonMath::Abs(currentCluster.phi - nextCluster.phi)};
            const float deltaZ{o2::gpu::GPUCommonMath::Abs(tanLambda * (nextCluster.radius - currentCluster.radius) + currentCluster.zCoordinate - nextCluster.zCoordinate)};

            if (deltaZ / sigmaZ < trkPars->NSigmaCut && (deltaPhi < phiCut || o2::gpu::GPUCommonMath::Abs(deltaPhi - constants::math::TwoPi) < phiCut)) {
              trackletsLookUpTable[currentSortedIndex]++; // Race-condition safe
              const float phi{o2::gpu::GPUCommonMath::ATan2(currentCluster.yCoordinate - nextCluster.yCoordinate, currentCluster.xCoordinate - nextCluster.xCoordinate)};
              const float tanL{(currentCluster.zCoordinate - nextCluster.zCoordinate) / (currentCluster.radius - nextCluster.radius)};
              const unsigned int stride{currentClusterIndex * maxTrackletsPerCluster};
              new (tracklets + stride + storedTracklets) Tracklet{currentSortedIndex, roFrameClustersNext[rof1] + iNextCluster, tanL, phi, rof0, rof1};
              ++storedTracklets;
            }
          }
        }
      }
    }
    // if (storedTracklets > maxTrackletsPerCluster) {
    //   printf("its-gpu-tracklet finder: found more tracklets per clusters (%d) than maximum set (%d), check the configuration!\n", maxTrackletsPerCluster, storedTracklets);
    // }
  }
}

template <int nLayers = 7>
GPUg() void compileTrackletsLookupTableKernel(const Tracklet* tracklets,
                                              int* trackletsLookUpTable,
                                              const int nTracklets)
{
  for (int currentTrackletIndex = blockIdx.x * blockDim.x + threadIdx.x; currentTrackletIndex < nTracklets; currentTrackletIndex += blockDim.x * gridDim.x) {
    auto& tracklet{tracklets[currentTrackletIndex]};
    if (tracklet.firstClusterIndex >= 0) {
      atomicAdd(trackletsLookUpTable + tracklet.firstClusterIndex, 1);
    }
  }
}

template <int nLayers = 7>
GPUg() void computeLayerTrackletsKernelMultipleRof(
  const int layerIndex,
  const int iteration,
  const unsigned int startRofId,
  const unsigned int rofSize,
  const int maxRofs,
  const Cluster* clustersCurrentLayer,        // input data rof0
  const Cluster* clustersNextLayer,           // input data rof0-delta <rof0< rof0+delta (up to 3 rofs)
  const int* roFrameClustersCurrentLayer,     // Number of clusters on layer 0 per ROF
  const int* roFrameClustersNextLayer,        // Number of clusters on layer 1 per ROF
  const int* indexTablesNext,                 // input data rof0-delta <rof0< rof0+delta (up to 3 rofs)
  const unsigned char* usedClustersLayer,     // input data rof0
  const unsigned char* usedClustersNextLayer, // input data rof1
  Tracklet* tracklets,                        // output data
  const Vertex* vertices,
  const int* nVertices,
  const float phiCut,
  const float minR,
  const float maxR,
  const float meanDeltaR,
  const float positionResolution,
  const float mSAngle,
  const StaticTrackingParameters<nLayers>* trkPars,
  const IndexTableUtils* utils,
  const unsigned int maxClustersPerRof = 5e2,
  const unsigned int maxTrackletsPerCluster = 50)
{
  const int phiBins{utils->getNphiBins()};
  const int zBins{utils->getNzBins()};
  for (unsigned int iRof{blockIdx.x}; iRof < rofSize; iRof += gridDim.x) {
    auto rof0 = iRof + startRofId;
    auto nClustersCurrentLayerRof = o2::gpu::GPUCommonMath::Min(roFrameClustersCurrentLayer[rof0 + 1] - roFrameClustersCurrentLayer[rof0], (int)maxClustersPerRof);
    // if (nClustersCurrentLayerRof > maxClustersPerRof) {
    //   printf("its-gpu-tracklet finder: on layer %d found more clusters per ROF (%d) than maximum set (%d), check the configuration!\n", layerIndex, nClustersCurrentLayerRof, maxClustersPerRof);
    // }
    auto* clustersCurrentLayerRof = clustersCurrentLayer + (roFrameClustersCurrentLayer[rof0] - roFrameClustersCurrentLayer[startRofId]);
    auto nVerticesRof0 = nVertices[rof0 + 1] - nVertices[rof0];
    auto trackletsRof0 = tracklets + maxTrackletsPerCluster * maxClustersPerRof * iRof;
    for (int currentClusterIndex = threadIdx.x; currentClusterIndex < nClustersCurrentLayerRof; currentClusterIndex += blockDim.x) {
      unsigned int storedTracklets{0};
      const Cluster& currentCluster{clustersCurrentLayerRof[currentClusterIndex]};
      const int currentSortedIndex{roFrameClustersCurrentLayer[rof0] + currentClusterIndex};
      const int currentSortedIndexChunk{currentSortedIndex - roFrameClustersCurrentLayer[startRofId]};
      if (usedClustersLayer[currentSortedIndex]) {
        continue;
      }

      int minRof = (rof0 >= trkPars->DeltaROF) ? rof0 - trkPars->DeltaROF : 0;
      int maxRof = (rof0 == maxRofs - trkPars->DeltaROF) ? rof0 : rof0 + trkPars->DeltaROF; // works with delta = {0, 1}
      const float inverseR0{1.f / currentCluster.radius};

      for (int iPrimaryVertex{0}; iPrimaryVertex < nVerticesRof0; iPrimaryVertex++) {
        const auto& primaryVertex{vertices[nVertices[rof0] + iPrimaryVertex]};
        const float resolution{o2::gpu::GPUCommonMath::Sqrt(Sq(trkPars->PVres) / primaryVertex.getNContributors() + Sq(positionResolution))};
        const float tanLambda{(currentCluster.zCoordinate - primaryVertex.getZ()) * inverseR0};
        const float zAtRmin{tanLambda * (minR - currentCluster.radius) + currentCluster.zCoordinate};
        const float zAtRmax{tanLambda * (maxR - currentCluster.radius) + currentCluster.zCoordinate};
        const float sqInverseDeltaZ0{1.f / (Sq(currentCluster.zCoordinate - primaryVertex.getZ()) + 2.e-8f)}; /// protecting from overflows adding the detector resolution
        const float sigmaZ{o2::gpu::CAMath::Sqrt(Sq(resolution) * Sq(tanLambda) * ((Sq(inverseR0) + sqInverseDeltaZ0) * Sq(meanDeltaR) + 1.f) + Sq(meanDeltaR * mSAngle))};

        const int4 selectedBinsRect{getBinsRect(currentCluster, layerIndex, *utils, zAtRmin, zAtRmax, sigmaZ * trkPars->NSigmaCut, phiCut)};

        if (selectedBinsRect.x == 0 && selectedBinsRect.y == 0 && selectedBinsRect.z == 0 && selectedBinsRect.w == 0) {
          continue;
        }
        int phiBinsNum{selectedBinsRect.w - selectedBinsRect.y + 1};
        if (phiBinsNum < 0) {
          phiBinsNum += trkPars->PhiBins;
        }
        const int tableSize{phiBins * zBins + 1};
        for (int rof1{minRof}; rof1 <= maxRof; ++rof1) {
          auto nClustersNext{roFrameClustersNextLayer[rof1 + 1] - roFrameClustersNextLayer[rof1]};
          if (!nClustersNext) { // number of clusters on next layer > 0
            continue;
          }
          for (int iPhiCount{0}; iPhiCount < phiBinsNum; iPhiCount++) {
            int iPhiBin = (selectedBinsRect.y + iPhiCount) % trkPars->PhiBins;
            const int firstBinIndex{utils->getBinIndex(selectedBinsRect.x, iPhiBin)};
            const int maxBinIndex{firstBinIndex + selectedBinsRect.z - selectedBinsRect.x + 1};
            const int firstRowClusterIndex = indexTablesNext[(rof1 - startRofId) * tableSize + firstBinIndex];
            const int maxRowClusterIndex = indexTablesNext[(rof1 - startRofId) * tableSize + maxBinIndex];
            for (int iNextCluster{firstRowClusterIndex}; iNextCluster < maxRowClusterIndex; ++iNextCluster) {
              if (iNextCluster >= nClustersNext) {
                break;
              }
              auto nextClusterIndex{roFrameClustersNextLayer[rof1] - roFrameClustersNextLayer[startRofId] + iNextCluster};
              const Cluster& nextCluster{clustersNextLayer[nextClusterIndex]};
              if (usedClustersNextLayer[nextCluster.clusterId]) {
                continue;
              }
              const float deltaPhi{o2::gpu::GPUCommonMath::Abs(currentCluster.phi - nextCluster.phi)};
              const float deltaZ{o2::gpu::GPUCommonMath::Abs(tanLambda * (nextCluster.radius - currentCluster.radius) + currentCluster.zCoordinate - nextCluster.zCoordinate)};

              if ((deltaZ / sigmaZ < trkPars->NSigmaCut && (deltaPhi < phiCut || o2::gpu::GPUCommonMath::Abs(deltaPhi - constants::math::TwoPi) < phiCut))) {
                const float phi{o2::gpu::GPUCommonMath::ATan2(currentCluster.yCoordinate - nextCluster.yCoordinate, currentCluster.xCoordinate - nextCluster.xCoordinate)};
                const float tanL{(currentCluster.zCoordinate - nextCluster.zCoordinate) / (currentCluster.radius - nextCluster.radius)};
                const unsigned int stride{currentClusterIndex * maxTrackletsPerCluster};
                if (storedTracklets < maxTrackletsPerCluster) {
                  new (trackletsRof0 + stride + storedTracklets) Tracklet{currentSortedIndexChunk, nextClusterIndex, tanL, phi, static_cast<short>(rof0), static_cast<short>(rof1)};
                }
                // else {
                // printf("its-gpu-tracklet-finder: on rof %d layer: %d: found more tracklets (%d) than maximum allowed per cluster. This is lossy!\n", rof0, layerIndex, storedTracklets);
                // }
                ++storedTracklets;
              }
            }
          }
        }
      }
    }
  }
}

// Decrease LUT entries corresponding to duplicated tracklets. NB: duplicate tracklets are removed separately (see const Tracklets*).
GPUg() void removeDuplicateTrackletsEntriesLUTKernel(
  int* trackletsLookUpTable,
  const Tracklet* tracklets,
  const int* nTracklets,
  const int layerIndex)
{
  int id0{-1}, id1{-1};
  for (int iTracklet{0}; iTracklet < nTracklets[layerIndex]; ++iTracklet) {
    auto& trk = tracklets[iTracklet];
    if (trk.firstClusterIndex == id0 && trk.secondClusterIndex == id1) {
      trackletsLookUpTable[id0]--;
    } else {
      id0 = trk.firstClusterIndex;
      id1 = trk.secondClusterIndex;
    }
  }
}

} // namespace gpu

void countCellsHandler(
  const Cluster** sortedClusters,
  const Cluster** unsortedClusters,
  const TrackingFrameInfo** tfInfo,
  const Tracklet** tracklets,
  const int** trackletsLUT,
  const int nTracklets,
  const int layer,
  CellSeed* cells,
  int** cellsLUTsArrayDevice,
  int* cellsLUTsHost,
  const float bz,
  const float maxChi2ClusterAttachment,
  const float cellDeltaTanLambdaSigma,
  const float nSigmaCut,
  const int nBlocks,
  const int nThreads)
{
  gpu::computeLayerCellsKernel<true><<<nBlocks, nThreads>>>(
    sortedClusters,           // const Cluster**
    unsortedClusters,         // const Cluster**
    tfInfo,                   // const TrackingFrameInfo**
    tracklets,                // const Tracklets**
    trackletsLUT,             // const int**
    nTracklets,               // const int
    layer,                    // const int
    cells,                    // CellSeed*
    cellsLUTsArrayDevice,     // int**
    bz,                       // const float
    maxChi2ClusterAttachment, // const float
    cellDeltaTanLambdaSigma,  // const float
    nSigmaCut);               // const float
  void* d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,     // d_temp_storage
                                              temp_storage_bytes, // temp_storage_bytes
                                              cellsLUTsHost,      // d_in
                                              cellsLUTsHost,      // d_out
                                              nTracklets + 1,     // num_items
                                              0));
  discardResult(hipMalloc(&d_temp_storage, temp_storage_bytes));
  gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage,     // d_temp_storage
                                              temp_storage_bytes, // temp_storage_bytes
                                              cellsLUTsHost,      // d_in
                                              cellsLUTsHost,      // d_out
                                              nTracklets + 1,     // num_items
                                              0));
  // gpu::printBufferLayerOnThread<<<1, 1>>>(layer, cellsLUTsHost, nTracklets + 1);
  gpuCheckError(hipFree(d_temp_storage));
}

void computeCellsHandler(
  const Cluster** sortedClusters,
  const Cluster** unsortedClusters,
  const TrackingFrameInfo** tfInfo,
  const Tracklet** tracklets,
  const int** trackletsLUT,
  const int nTracklets,
  const int layer,
  CellSeed* cells,
  int** cellsLUTsArrayDevice,
  int* cellsLUTsHost,
  const float bz,
  const float maxChi2ClusterAttachment,
  const float cellDeltaTanLambdaSigma,
  const float nSigmaCut,
  const int nBlocks,
  const int nThreads)
{
  gpu::computeLayerCellsKernel<false><<<nBlocks, nThreads>>>(
    sortedClusters,           // const Cluster**
    unsortedClusters,         // const Cluster**
    tfInfo,                   // const TrackingFrameInfo**
    tracklets,                // const Tracklets**
    trackletsLUT,             // const int**
    nTracklets,               // const int
    layer,                    // const int
    cells,                    // CellSeed*
    cellsLUTsArrayDevice,     // int**
    bz,                       // const float
    maxChi2ClusterAttachment, // const float
    cellDeltaTanLambdaSigma,  // const float
    nSigmaCut);               // const float
}

void countCellNeighboursHandler(CellSeed** cellsLayersDevice,
                                int* neighboursLUT,
                                int** cellsLUTs,
                                gpuPair<int, int>* cellNeighbours,
                                int* neighboursIndexTable,
                                const float maxChi2ClusterAttachment,
                                const float bz,
                                const int layerIndex,
                                const unsigned int nCells,
                                const unsigned int nCellsNext,
                                const int maxCellNeighbours,
                                const int nBlocks,
                                const int nThreads)
{
  gpu::computeLayerCellNeighboursKernel<true><<<nBlocks, nThreads>>>(
    cellsLayersDevice,
    neighboursLUT,
    neighboursIndexTable,
    cellsLUTs,
    cellNeighbours,
    maxChi2ClusterAttachment,
    bz,
    layerIndex,
    nCells,
    maxCellNeighbours);
  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
  void *d_temp_storage = nullptr, *d_temp_storage_2 = nullptr;
  size_t temp_storage_bytes = 0, temp_storage_bytes_2 = 0;
  gpuCheckError(hipcub::DeviceScan::InclusiveSum(d_temp_storage,     // d_temp_storage
                                              temp_storage_bytes, // temp_storage_bytes
                                              neighboursLUT,      // d_in
                                              neighboursLUT,      // d_out
                                              nCellsNext));       // num_items

  discardResult(hipMalloc(&d_temp_storage, temp_storage_bytes));
  gpuCheckError(hipcub::DeviceScan::InclusiveSum(d_temp_storage,       // d_temp_storage
                                              temp_storage_bytes,   // temp_storage_bytes
                                              neighboursLUT,        // d_in
                                              neighboursLUT,        // d_out
                                              nCellsNext));         // num_items
  gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage_2,     // d_temp_storage
                                              temp_storage_bytes_2, // temp_storage_bytes
                                              neighboursIndexTable, // d_in
                                              neighboursIndexTable, // d_out
                                              nCells + 1,           // num_items
                                              0));
  discardResult(hipMalloc(&d_temp_storage_2, temp_storage_bytes_2));
  gpuCheckError(hipcub::DeviceScan::ExclusiveSum(d_temp_storage_2,     // d_temp_storage
                                              temp_storage_bytes_2, // temp_storage_bytes
                                              neighboursIndexTable, // d_in
                                              neighboursIndexTable, // d_out
                                              nCells + 1,           // num_items
                                              0));
  gpuCheckError(hipFree(d_temp_storage));
  gpuCheckError(hipFree(d_temp_storage_2));
  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
}

void computeCellNeighboursHandler(CellSeed** cellsLayersDevice,
                                  int* neighboursLUT,
                                  int** cellsLUTs,
                                  gpuPair<int, int>* cellNeighbours,
                                  int* neighboursIndexTable,
                                  const float maxChi2ClusterAttachment,
                                  const float bz,
                                  const int layerIndex,
                                  const unsigned int nCells,
                                  const unsigned int nCellsNext,
                                  const int maxCellNeighbours,
                                  const int nBlocks,
                                  const int nThreads)
{

  gpu::computeLayerCellNeighboursKernel<false><<<o2::gpu::GPUCommonMath::Min(nBlocks, GPU_BLOCKS),
                                                 o2::gpu::GPUCommonMath::Min(nThreads, GPU_THREADS)>>>(
    cellsLayersDevice,
    neighboursLUT,
    neighboursIndexTable,
    cellsLUTs,
    cellNeighbours,
    maxChi2ClusterAttachment,
    bz,
    layerIndex,
    nCells,
    maxCellNeighbours);
  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
}

void filterCellNeighboursHandler(std::vector<int>& neighHost,
                                 gpuPair<int, int>* cellNeighbours,
                                 unsigned int nNeigh)
{
  thrust::device_ptr<gpuPair<int, int>> neighVector(cellNeighbours);
  thrust::device_vector<int> keys(nNeigh); // TODO: externally allocate.
  thrust::device_vector<int> vals(nNeigh); // TODO: externally allocate.
  thrust::copy(thrust::make_transform_iterator(neighVector, gpu::pair_to_second<int, int>()),
               thrust::make_transform_iterator(neighVector + nNeigh, gpu::pair_to_second<int, int>()),
               keys.begin());
  thrust::sequence(vals.begin(), vals.end());
  thrust::sort_by_key(keys.begin(), keys.end(), vals.begin());
  thrust::device_vector<gpuPair<int, int>> sortedNeigh(nNeigh);
  thrust::copy(thrust::make_permutation_iterator(neighVector, vals.begin()),
               thrust::make_permutation_iterator(neighVector, vals.end()),
               sortedNeigh.begin());
  discardResult(hipDeviceSynchronize());
  auto trimmedBegin = thrust::find_if(sortedNeigh.begin(), sortedNeigh.end(), gpu::is_valid_pair<int, int>()); // trim leading -1s
  auto trimmedSize = sortedNeigh.end() - trimmedBegin;
  thrust::device_vector<int> validNeigh(trimmedSize);
  neighHost.resize(trimmedSize);
  thrust::transform(trimmedBegin, sortedNeigh.end(), validNeigh.begin(), gpu::pair_to_first<int, int>());
  gpuCheckError(hipMemcpy(neighHost.data(), thrust::raw_pointer_cast(validNeigh.data()), trimmedSize * sizeof(int), hipMemcpyDeviceToHost));
}

void trackSeedHandler(CellSeed* trackSeeds,
                      const TrackingFrameInfo** foundTrackingFrameInfo,
                      o2::its::TrackITSExt* tracks,
                      const unsigned int nSeeds,
                      const float Bz,
                      const int startLevel,
                      float maxChi2ClusterAttachment,
                      float maxChi2NDF,
                      const o2::base::Propagator* propagator,
                      const o2::base::PropagatorF::MatCorrType matCorrType,
                      const int nBlocks,
                      const int nThreads)
{
  gpu::fitTrackSeedsKernel<<<nBlocks, nThreads>>>(
    trackSeeds,               // CellSeed*
    foundTrackingFrameInfo,   // TrackingFrameInfo**
    tracks,                   // TrackITSExt*
    nSeeds,                   // const unsigned int
    Bz,                       // const float
    startLevel,               // const int
    maxChi2ClusterAttachment, // float
    maxChi2NDF,               // float
    propagator,               // const o2::base::Propagator*
    matCorrType);             // o2::base::PropagatorF::MatCorrType

  gpuCheckError(hipPeekAtLastError());
  gpuCheckError(hipDeviceSynchronize());
}
} // namespace o2::its