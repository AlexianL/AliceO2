#include "hip/hip_runtime.h"
// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.

#include <hip/hip_runtime.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>

#include "ITStracking/Constants.h"

#include "ITStrackingGPU/Utils.h"
#include "ITStrackingGPU/TracerGPU.h"

#include "ITStrackingGPU/TimeFrameChunk.h"

#include <unistd.h>
#include <thread>

#include "GPUCommonDef.h"
#include "GPUCommonMath.h"
#include "GPUCommonLogger.h"

#ifndef __HIPCC__
#define THRUST_NAMESPACE thrust::cuda
#else
#define THRUST_NAMESPACE thrust::hip
#endif

namespace o2::its
{
using constants::GB;
using constants::MB;
namespace gpu
{
using utils::checkGPUError;

template <int nLayers>
GpuTimeFrameChunk<nLayers>::~GpuTimeFrameChunk()
{
  if (mAllocated) {
    for (int i = 0; i < nLayers; ++i) {
      checkGPUError(hipFree(mClustersDevice[i]));
      // checkGPUError(hipFree(mTrackingFrameInfoDevice[i]));
      checkGPUError(hipFree(mClusterExternalIndicesDevice[i]));
      checkGPUError(hipFree(mIndexTablesDevice[i]));
      if (i < nLayers - 1) {
        checkGPUError(hipFree(mTrackletsDevice[i]));
        checkGPUError(hipFree(mTrackletsLookupTablesDevice[i]));
        if (i < nLayers - 2) {
          checkGPUError(hipFree(mCellsDevice[i]));
          checkGPUError(hipFree(mCellsLookupTablesDevice[i]));
          checkGPUError(hipFree(mRoadsLookupTablesDevice[i]));
          if (i < nLayers - 3) {
            checkGPUError(hipFree(mNeighboursCellLookupTablesDevice[i]));
            checkGPUError(hipFree(mNeighboursCellDevice[i]));
          }
        }
      }
    }
    // checkGPUError(hipFree(mRoadsDevice));
    checkGPUError(hipFree(mCUBTmpBufferDevice));
    checkGPUError(hipFree(mFoundTrackletsDevice));
    checkGPUError(hipFree(mNFoundCellsDevice));
    checkGPUError(hipFree(mCellsDeviceArray));
    checkGPUError(hipFree(mNeighboursCellDeviceArray));
    checkGPUError(hipFree(mNeighboursCellLookupTablesDeviceArray));
  }
}

template <int nLayers>
void GpuTimeFrameChunk<nLayers>::allocate(const size_t nrof, Stream& stream)
{
  RANGE("device_partition_allocation", 2);
  mNRof = nrof;
  // for (int i = 0; i < nLayers; ++i) {
  //   static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mClustersDevice[i])), sizeof(Cluster) * mTFGPUParams->clustersPerROfCapacity * nrof, &stream, true);
  //   // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mTrackingFrameInfoDevice[i])), sizeof(TrackingFrameInfo) * mTFGPUParams->clustersPerROfCapacity * nrof, &stream, true);
  //   static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mClusterExternalIndicesDevice[i])), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * nrof, &stream, true);
  //   static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mIndexTablesDevice[i])), sizeof(int) * (256 * 128 + 1) * nrof, &stream, true);
  //   if (i < nLayers - 1) {
  //     static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mTrackletsLookupTablesDevice[i])), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * nrof, &stream, true);
  //     static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mTrackletsDevice[i])), sizeof(Tracklet) * mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * nrof, &stream, true);
  //     if (i < nLayers - 2) {
  //       static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mCellsLookupTablesDevice[i])), sizeof(int) * mTFGPUParams->validatedTrackletsCapacity * nrof, &stream, true);
  //       static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mCellsDevice[i])), sizeof(CellSeed) * mTFGPUParams->maxNeighboursSize * nrof, &stream, true);
  //       static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mRoadsLookupTablesDevice[i]), sizeof(int) * mTFGPUParams->maxNeighboursSize * nrof, &stream, true);
  //       if (i < nLayers - 3) {
  //         static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mNeighboursCellLookupTablesDevice[i])), sizeof(int) * mTFGPUParams->maxNeighboursSize * nrof, &stream, true);
  //         static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mNeighboursCellDevice[i])), sizeof(int) * mTFGPUParams->maxNeighboursSize * nrof, &stream, true);
  //       }
  //       if (i < 2) {
  //         static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&(mNTrackletsPerClusterDevice[i])), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * nrof, &stream, true);
  //       }
  //     }
  //   }
  // }
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mCUBTmpBufferDevice), mTFGPUParams->tmpCUBBufferSize * nrof, &stream, true);
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mLinesDevice), sizeof(Line) * mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * nrof, &stream, true);
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mNFoundLinesDevice), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * nrof, &stream, true);
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mNExclusiveFoundLinesDevice), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * nrof + 1, &stream, true); // + 1 for hipcub::DeviceScan::ExclusiveSum, to cover cases where we have maximum number of clusters per ROF
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mUsedTrackletsDevice), sizeof(unsigned char) * mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * nrof, &stream, true);
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mClusteredLinesDevice), sizeof(int) * mTFGPUParams->clustersPerROfCapacity * mTFGPUParams->maxTrackletsPerCluster * nrof, &stream, true);
  // // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mRoadsDevice), sizeof(Road<nLayers - 2>) * mTFGPUParams->maxRoadPerRofSize * nrof, &stream, true);

  // /// Invariant allocations
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mFoundTrackletsDevice), (nLayers - 1) * sizeof(int) * nrof, &stream, true); // No need to reset, we always read it after writing
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mNFoundCellsDevice), (nLayers - 2) * sizeof(int) * nrof, &stream, true);
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mCellsDeviceArray), (nLayers - 2) * sizeof(CellSeed*), &stream, true);
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mNeighboursCellDeviceArray), (nLayers - 3) * sizeof(int*), &stream, true);
  // static_cast<TimeFrameGPU<nLayers>*>(mTimeFramePtr)->allocMemAsync(reinterpret_cast<void**>(&mNeighboursCellLookupTablesDeviceArray), (nLayers - 3) * sizeof(int*), &stream, true);

  // /// Copy pointers of allocated memory to regrouping arrays
  // checkGPUError(hipMemcpyAsync(mCellsDeviceArray, mCellsDevice.data(), (nLayers - 2) * sizeof(CellSeed*), hipMemcpyHostToDevice, stream.get()));
  // checkGPUError(hipMemcpyAsync(mNeighboursCellDeviceArray, mNeighboursCellDevice.data(), (nLayers - 3) * sizeof(int*), hipMemcpyHostToDevice, stream.get()));
  // checkGPUError(hipMemcpyAsync(mNeighboursCellLookupTablesDeviceArray, mNeighboursCellLookupTablesDevice.data(), (nLayers - 3) * sizeof(int*), hipMemcpyHostToDevice, stream.get()));

  mAllocated = true;
}

template <int nLayers>
void GpuTimeFrameChunk<nLayers>::reset(const Task task, Stream& stream)
{
  RANGE("buffer_reset", 0);
  // if ((bool)task) { // Vertexer-only initialisation (cannot be constexpr: due to the presence of gpu raw calls can't be put in header)
  //   for (int i = 0; i < 2; i++) {
  //     auto thrustTrackletsBegin = thrust::device_ptr<Tracklet>(mTrackletsDevice[i]);
  //     auto thrustTrackletsEnd = thrustTrackletsBegin + mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * mNRof;
  //     thrust::fill(THRUST_NAMESPACE::par.on(stream.get()), thrustTrackletsBegin, thrustTrackletsEnd, Tracklet{});
  //     checkGPUError(hipMemsetAsync(mNTrackletsPerClusterDevice[i], 0, sizeof(int) * mTFGPUParams->clustersPerROfCapacity * mNRof, stream.get()));
  //   }
  //   checkGPUError(hipMemsetAsync(mUsedTrackletsDevice, false, sizeof(unsigned char) * mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * mNRof, stream.get()));
  //   checkGPUError(hipMemsetAsync(mClusteredLinesDevice, -1, sizeof(int) * mTFGPUParams->clustersPerROfCapacity * mTFGPUParams->maxTrackletsPerCluster * mNRof, stream.get()));
  // } else {
  //   for (int i = 0; i < nLayers; ++i) {
  //     if (i < nLayers - 1) {
  //       checkGPUError(hipMemsetAsync(mTrackletsLookupTablesDevice[i], 0, sizeof(int) * mTFGPUParams->clustersPerROfCapacity * mNRof, stream.get()));
  //       auto thrustTrackletsBegin = thrust::device_ptr<Tracklet>(mTrackletsDevice[i]);
  //       auto thrustTrackletsEnd = thrustTrackletsBegin + mTFGPUParams->maxTrackletsPerCluster * mTFGPUParams->clustersPerROfCapacity * mNRof;
  //       thrust::fill(THRUST_NAMESPACE::par.on(stream.get()), thrustTrackletsBegin, thrustTrackletsEnd, Tracklet{});
  //       if (i < nLayers - 2) {
  //         checkGPUError(hipMemsetAsync(mCellsLookupTablesDevice[i], 0, sizeof(int) * mTFGPUParams->cellsLUTsize * mNRof, stream.get()));
  //         checkGPUError(hipMemsetAsync(mRoadsLookupTablesDevice[i], 0, sizeof(int) * mTFGPUParams->maxNeighboursSize * mNRof, stream.get()));
  //         if (i < nLayers - 3) {
  //           checkGPUError(hipMemsetAsync(mNeighboursCellLookupTablesDevice[i], 0, sizeof(int) * mTFGPUParams->maxNeighboursSize * mNRof, stream.get()));
  //           checkGPUError(hipMemsetAsync(mNeighboursCellDevice[i], 0, sizeof(int) * mTFGPUParams->maxNeighboursSize * mNRof, stream.get()));
  //         }
  //       }
  //     }
  //   }
  //   checkGPUError(hipMemsetAsync(mNFoundCellsDevice, 0, (nLayers - 2) * sizeof(int), stream.get()));
  // }
}

template <int nLayers>
size_t GpuTimeFrameChunk<nLayers>::computeScalingSizeBytes(const int nrof, const TimeFrameGPUParameters& config)
{
  size_t rofsize = nLayers * sizeof(int); // number of clusters per ROF
  // rofsize += nLayers * sizeof(Cluster) * config.clustersPerROfCapacity;                                        // clusters
  // rofsize += nLayers * sizeof(TrackingFrameInfo) * config.clustersPerROfCapacity;                              // tracking frame info
  // rofsize += nLayers * sizeof(int) * config.clustersPerROfCapacity;                                            // external cluster indices
  // rofsize += nLayers * sizeof(int) * (256 * 128 + 1);                                                          // index tables
  // rofsize += (nLayers - 1) * sizeof(int) * config.clustersPerROfCapacity;                                      // tracklets lookup tables
  // rofsize += (nLayers - 1) * sizeof(Tracklet) * config.maxTrackletsPerCluster * config.clustersPerROfCapacity; // tracklets
  // rofsize += 2 * sizeof(int) * config.clustersPerROfCapacity;                                                  // tracklets found per cluster (vertexer)
  // rofsize += sizeof(unsigned char) * config.maxTrackletsPerCluster * config.clustersPerROfCapacity;            // used tracklets (vertexer)
  // rofsize += (nLayers - 2) * sizeof(int) * config.validatedTrackletsCapacity;                                  // cells lookup tables
  // rofsize += (nLayers - 2) * sizeof(CellSeed) * config.maxNeighboursSize;                                      // cells
  // rofsize += (nLayers - 3) * sizeof(int) * config.maxNeighboursSize;                                           // cell neighbours lookup tables
  // rofsize += (nLayers - 3) * sizeof(int) * config.maxNeighboursSize;                                           // cell neighbours
  // rofsize += sizeof(Road<nLayers - 2>) * config.maxRoadPerRofSize;                                             // roads
  // rofsize += (nLayers - 2) * sizeof(int) * config.maxNeighboursSize;                                           // road LUT
  // rofsize += sizeof(Line) * config.maxTrackletsPerCluster * config.clustersPerROfCapacity;                     // lines
  // rofsize += sizeof(int) * config.clustersPerROfCapacity;                                                      // found lines
  // rofsize += sizeof(int) * config.clustersPerROfCapacity;                                                      // found lines exclusive sum
  // rofsize += sizeof(int) * config.clustersPerROfCapacity * config.maxTrackletsPerCluster;                      // lines used in clusterlines

  // rofsize += (nLayers - 1) * sizeof(int); // total found tracklets
  // rofsize += (nLayers - 2) * sizeof(int); // total found cells

  return rofsize * nrof;
}

template <int nLayers>
size_t GpuTimeFrameChunk<nLayers>::computeFixedSizeBytes(const TimeFrameGPUParameters& config)
{
  size_t total = config.tmpCUBBufferSize;                  // CUB tmp buffers
  total += sizeof(gpu::StaticTrackingParameters<nLayers>); // static parameters loaded once
  return total;
}

template <int nLayers>
size_t GpuTimeFrameChunk<nLayers>::computeRofPerChunk(const TimeFrameGPUParameters& config, const size_t m)
{
  return (m * GB / (float)(config.nTimeFrameChunks) - GpuTimeFrameChunk<nLayers>::computeFixedSizeBytes(config)) / (float)GpuTimeFrameChunk<nLayers>::computeScalingSizeBytes(1, config);
}

/// Interface
template <int nLayers>
Cluster* GpuTimeFrameChunk<nLayers>::getDeviceClusters(const int layer)
{
  return mClustersDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceClusterExternalIndices(const int layer)
{
  return mClusterExternalIndicesDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceIndexTables(const int layer)
{
  return mIndexTablesDevice[layer];
}

template <int nLayers>
Tracklet* GpuTimeFrameChunk<nLayers>::getDeviceTracklets(const int layer)
{
  return mTrackletsDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceTrackletsLookupTables(const int layer)
{
  return mTrackletsLookupTablesDevice[layer];
}

template <int nLayers>
CellSeed* GpuTimeFrameChunk<nLayers>::getDeviceCells(const int layer)
{
  return mCellsDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceCellsLookupTables(const int layer)
{
  return mCellsLookupTablesDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceCellNeigboursLookupTables(const int layer)
{
  return mNeighboursCellLookupTablesDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceCellNeighbours(const int layer)
{
  return mNeighboursCellDevice[layer];
}

template <int nLayers>
int* GpuTimeFrameChunk<nLayers>::getDeviceRoadsLookupTables(const int layer)
{
  return mRoadsLookupTablesDevice[layer];
}

// Load data
template <int nLayers>
size_t GpuTimeFrameChunk<nLayers>::loadDataOnDevice(const size_t startRof, const size_t maxRof, const int maxLayers, Stream& stream)
{
  RANGE("load_clusters_data", 5);
  // auto nRofs = std::min(maxRof - startRof, mNRof);
  // mNPopulatedRof = mTimeFramePtr->getNClustersROFrange(startRof, nRofs, 0).size();
  // for (int i = 0; i < maxLayers; ++i) {
  //   mHostClusters[i] = mTimeFramePtr->getClustersPerROFrange(startRof, nRofs, i);
  //   mHostIndexTables[i] = mTimeFramePtr->getIndexTablePerROFrange(startRof, nRofs, i);
  //   if (mHostClusters[i].size() > mTFGPUParams->clustersPerROfCapacity * nRofs) {
  //     LOGP(warning, "Clusters on layer {} exceed the expected value, resizing to config value: {}, will lose information!", i, mTFGPUParams->clustersPerROfCapacity * nRofs);
  //   }
  //   checkGPUError(hipMemcpyAsync(mClustersDevice[i],
  //                                 mHostClusters[i].data(),
  //                                 (int)std::min(mHostClusters[i].size(), mTFGPUParams->clustersPerROfCapacity * nRofs) * sizeof(Cluster),
  //                                 hipMemcpyHostToDevice, stream.get()));
  //   if (mHostIndexTables[i].data()) {
  //     checkGPUError(hipMemcpyAsync(mIndexTablesDevice[i],
  //                                   mHostIndexTables[i].data(),
  //                                   mHostIndexTables[i].size() * sizeof(int),
  //                                   hipMemcpyHostToDevice, stream.get()));
  //   }
  // }
  return mNPopulatedRof; // return the number of ROFs we loaded the data for.
}
template class GpuTimeFrameChunk<7>;
} // namespace gpu
} // namespace o2::its