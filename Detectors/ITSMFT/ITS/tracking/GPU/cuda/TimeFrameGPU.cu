// Copyright 2019-2020 CERN and copyright holders of ALICE O2.
// See https://alice-o2.web.cern.ch/copyright for details of the copyright holders.
// All rights not expressly granted are reserved.
//
// This software is distributed under the terms of the GNU General Public
// License v3 (GPL Version 3), copied verbatim in the file "COPYING".
//
// In applying this license CERN does not waive the privileges and immunities
// granted to it by virtue of its status as an Intergovernmental Organization
// or submit itself to any jurisdiction.
///
#include <hip/hip_runtime.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>

#include "ITStracking/Constants.h"

#include "ITStrackingGPU/Utils.h"
#include "ITStrackingGPU/TimeFrameGPU.h"
#include "ITStrackingGPU/TracerGPU.h"

#include <unistd.h>
#include <thread>

#include "GPUCommonDef.h"
#include "GPUCommonMath.h"
#include "GPUCommonLogger.h"

#ifndef __HIPCC__
#define THRUST_NAMESPACE thrust::cuda
#else
#define THRUST_NAMESPACE thrust::hip
#endif

namespace o2
{
namespace its
{
using constants::GB;
using constants::MB;

namespace gpu
{
using utils::checkGPUError;

void* DefaultGPUAllocator::allocate(size_t size)
{
  LOGP(fatal, "Called DefaultGPUAllocator::allocate with size {}", size);
  return nullptr; // to be implemented
}

template <int nLayers>
TimeFrameGPU<nLayers>::TimeFrameGPU()
{
  mIsGPU = true;
  utils::getDeviceProp(0, true);
}

template <int nLayers>
TimeFrameGPU<nLayers>::~TimeFrameGPU() = default;

template <int nLayers>
void TimeFrameGPU<nLayers>::allocMemAsync(void** ptr, size_t size, Stream* strPtr, bool extAllocator)
{
  if (extAllocator) {
    *ptr = mAllocator->allocate(size);
  } else {
    LOGP(info, "Calling default CUDA allocator");
    checkGPUError(hipMallocAsync(reinterpret_cast<void**>(ptr), size, strPtr->get()));
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::setDevicePropagator(const o2::base::PropagatorImpl<float>* propagator)
{
  mPropagatorDevice = propagator;
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadUnsortedClustersDevice()
{
  for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading {} unsorted clusters on layer {}, for {} MB.", mUnsortedClusters[iLayer].size(), iLayer, mUnsortedClusters[iLayer].size() * sizeof(Cluster) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mUnsortedClustersDevice[iLayer]), mUnsortedClusters[iLayer].size() * sizeof(Cluster), nullptr, getExtAllocator());
    // Register and move data
    checkGPUError(hipHostRegister(mUnsortedClusters[iLayer].data(), mUnsortedClusters[iLayer].size() * sizeof(Cluster), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mUnsortedClustersDevice[iLayer], mUnsortedClusters[iLayer].data(), mUnsortedClusters[iLayer].size() * sizeof(Cluster), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  allocMemAsync(reinterpret_cast<void**>(&mUnsortedClustersDeviceArray), nLayers * sizeof(Cluster*), nullptr, getExtAllocator());
  checkGPUError(hipHostRegister(mUnsortedClustersDevice.data(), nLayers * sizeof(Cluster*), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mUnsortedClustersDeviceArray, mUnsortedClustersDevice.data(), nLayers * sizeof(Cluster*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadClustersDevice()
{
  for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading {} clusters on layer {}, for {} MB.", mClusters[iLayer].size(), iLayer, mClusters[iLayer].size() * sizeof(Cluster) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mClustersDevice[iLayer]), mClusters[iLayer].size() * sizeof(Cluster), nullptr, getExtAllocator());
    // Register and move data
    checkGPUError(hipHostRegister(mClusters[iLayer].data(), mClusters[iLayer].size() * sizeof(Cluster), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mClustersDevice[iLayer], mClusters[iLayer].data(), mClusters[iLayer].size() * sizeof(Cluster), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  allocMemAsync(reinterpret_cast<void**>(&mClustersDeviceArray), nLayers * sizeof(Cluster*), nullptr, getExtAllocator());
  checkGPUError(hipHostRegister(mClustersDevice.data(), nLayers * sizeof(Cluster*), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mClustersDeviceArray, mClustersDevice.data(), nLayers * sizeof(Cluster*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadTrackingFrameInfoDevice(const int iteration)
{
  if (!iteration) {
    for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
      LOGP(debug, "gpu-transfer: loading {} tfinfo on layer {}, for {} MB.", mTrackingFrameInfo[iLayer].size(), iLayer, mTrackingFrameInfo[iLayer].size() * sizeof(TrackingFrameInfo) / MB);
      allocMemAsync(reinterpret_cast<void**>(&mTrackingFrameInfoDevice[iLayer]), mTrackingFrameInfo[iLayer].size() * sizeof(TrackingFrameInfo), nullptr, getExtAllocator());
      // Register and move data
      checkGPUError(hipHostRegister(mTrackingFrameInfo[iLayer].data(), mTrackingFrameInfo[iLayer].size() * sizeof(TrackingFrameInfo), hipHostRegisterPortable));
      checkGPUError(hipMemcpyAsync(mTrackingFrameInfoDevice[iLayer], mTrackingFrameInfo[iLayer].data(), mTrackingFrameInfo[iLayer].size() * sizeof(TrackingFrameInfo), hipMemcpyHostToDevice, mGpuStreams[0].get()));
    }
    allocMemAsync(reinterpret_cast<void**>(&mTrackingFrameInfoDeviceArray), nLayers * sizeof(TrackingFrameInfo*), nullptr, getExtAllocator());
    checkGPUError(hipHostRegister(mTrackingFrameInfoDevice.data(), nLayers * sizeof(TrackingFrameInfo*), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mTrackingFrameInfoDeviceArray, mTrackingFrameInfoDevice.data(), nLayers * sizeof(TrackingFrameInfo*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadTrackletsDevice()
{
  for (auto iLayer{0}; iLayer < nLayers - 1; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading {} tracklets on layer {}, for {} MB.", mTracklets[iLayer].size(), iLayer, mTracklets[iLayer].size() * sizeof(Tracklet) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mTrackletsDevice[iLayer]), mTracklets[iLayer].size() * sizeof(Tracklet), nullptr, getExtAllocator());
    // Register and move data
    checkGPUError(hipHostRegister(mTracklets[iLayer].data(), mTracklets[iLayer].size() * sizeof(Tracklet), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mTrackletsDevice[iLayer], mTracklets[iLayer].data(), mTracklets[iLayer].size() * sizeof(Tracklet), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  allocMemAsync(reinterpret_cast<void**>(&mTrackletsDeviceArray), (nLayers - 1) * sizeof(Tracklet*), nullptr, getExtAllocator());
  checkGPUError(hipHostRegister(mTrackletsDevice.data(), (nLayers - 1) * sizeof(Tracklet*), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mTrackletsDeviceArray, mTrackletsDevice.data(), (nLayers - 1) * sizeof(Tracklet*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadCellsDevice()
{
  for (auto iLayer{0}; iLayer < nLayers - 2; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading {} cell seeds on layer {}, for {} MB.", mCells[iLayer].size(), iLayer, mCells[iLayer].size() * sizeof(CellSeed) / MB);
    allocMemAsync(reinterpret_cast<void**>(&mCellsDevice[iLayer]), mCells[iLayer].size() * sizeof(CellSeed), nullptr, getExtAllocator());
    allocMemAsync(reinterpret_cast<void**>(&mNeighboursIndexTablesDevice[iLayer]), (mCells[iLayer].size() + 1) * sizeof(int), nullptr, getExtAllocator()); // accessory for the neigh. finding.
    checkGPUError(hipMemsetAsync(mNeighboursIndexTablesDevice[iLayer], 0, (mCells[iLayer].size() + 1) * sizeof(int), mGpuStreams[0].get()));
    // Register and move data
    checkGPUError(hipHostRegister(mCells[iLayer].data(), mCells[iLayer].size() * sizeof(CellSeed), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mCellsDevice[iLayer], mCells[iLayer].data(), mCells[iLayer].size() * sizeof(CellSeed), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  allocMemAsync(reinterpret_cast<void**>(&mCellsDeviceArray), (nLayers - 2) * sizeof(CellSeed*), nullptr, getExtAllocator());
  checkGPUError(hipHostRegister(mCellsDevice.data(), (nLayers - 2) * sizeof(CellSeed*), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mCellsDeviceArray, mCellsDevice.data(), (nLayers - 2) * sizeof(CellSeed*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadCellsLUT()
{
  for (auto iLayer{0}; iLayer < nLayers - 3; ++iLayer) {
    LOGP(debug, "gpu-transfer: loading {} cell LUTs on layer {}, for {} MB.", mCellsLookupTable[iLayer].size(), iLayer, mCellsLookupTable[iLayer].size() * sizeof(int) / MB);
    allocMemAsync(reinterpret_cast<void**>(&(mCellsLUTDevice[iLayer])), sizeof(int) * mCellsLookupTable[iLayer].size(), nullptr, getExtAllocator());
    // Register and move data
    checkGPUError(hipHostRegister(mCellsLookupTable[iLayer].data(), mCellsLookupTable[iLayer].size() * sizeof(int), hipHostRegisterPortable));
    checkGPUError(hipMemcpyAsync(mCellsLUTDevice[iLayer], mCellsLookupTable[iLayer].data(), mCellsLookupTable[iLayer].size() * sizeof(int), hipMemcpyHostToDevice, mGpuStreams[0].get()));
  }
  allocMemAsync(reinterpret_cast<void**>(&mCellsLUTDeviceArray), (nLayers - 2) * sizeof(int*), nullptr, getExtAllocator());
  checkGPUError(hipHostRegister(mCellsLUTDevice.data(), mCellsLUTDevice.size() * sizeof(int*), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mCellsLUTDeviceArray, mCellsLUTDevice.data(), mCellsLUTDevice.size() * sizeof(int*), hipMemcpyHostToDevice, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadRoadsDevice()
{
  LOGP(debug, "gpu-transfer: loading {} roads, for {} MB.", mRoads.size(), mRoads.size() * sizeof(Road<nLayers - 2>) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mRoadsDevice), mRoads.size() * sizeof(Road<nLayers - 2>), &(mGpuStreams[0]), getExtAllocator());
  checkGPUError(hipHostRegister(mRoads.data(), mRoads.size() * sizeof(Road<nLayers - 2>), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mRoadsDevice, mRoads.data(), mRoads.size() * sizeof(Road<nLayers - 2>), hipMemcpyHostToDevice, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::loadTrackSeedsDevice(std::vector<CellSeed>& seeds)
{
  LOGP(debug, "gpu-transfer: loading {} track seeds, for {} MB.", seeds.size(), seeds.size() * sizeof(CellSeed) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mTrackSeedsDevice), seeds.size() * sizeof(CellSeed), &(mGpuStreams[0]), getExtAllocator());
  checkGPUError(hipHostRegister(seeds.data(), seeds.size() * sizeof(CellSeed), hipHostRegisterPortable));
  checkGPUError(hipMemcpyAsync(mTrackSeedsDevice, seeds.data(), seeds.size() * sizeof(CellSeed), hipMemcpyHostToDevice, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createNeighboursDevice(const unsigned int& layer, std::vector<std::pair<int, int>>& neighbours)
{
  mCellsNeighbours[layer].clear();
  mCellsNeighbours[layer].resize(neighbours.size());
  LOGP(debug, "gpu-allocation: reserving {} neighbours, for {} MB.", neighbours.size(), neighbours.size() * sizeof(gpuPair<int, int>) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mNeighboursDevice[layer]), neighbours.size() * sizeof(gpuPair<int, int>), &(mGpuStreams[0]), getExtAllocator());
  checkGPUError(hipMemsetAsync(mNeighboursDevice[layer], -1, neighbours.size() * sizeof(gpuPair<int, int>), mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createNeighboursLUTDevice(const int layer, const unsigned int nCells)
{
  LOGP(debug, "gpu-allocation: reserving {} slots for neighbours LUT, for {} MB.", nCells + 1, (nCells + 1) * sizeof(int) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mNeighboursLUTDevice[layer]), (nCells + 1) * sizeof(int), nullptr, getExtAllocator()); // We need one element more to move exc -> inc
  checkGPUError(hipMemsetAsync(mNeighboursLUTDevice[layer], 0, (nCells + 1) * sizeof(int), mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::createTrackITSExtDevice(std::vector<CellSeed>& seeds)
{
  mTrackITSExt.clear();
  mTrackITSExt.resize(seeds.size());
  LOGP(debug, "gpu-allocation: reserving {} tracks, for {} MB.", seeds.size(), seeds.size() * sizeof(o2::its::TrackITSExt) / MB);
  allocMemAsync(reinterpret_cast<void**>(&mTrackITSExtDevice), seeds.size() * sizeof(o2::its::TrackITSExt), &(mGpuStreams[0]), getExtAllocator());
  checkGPUError(hipMemsetAsync(mTrackITSExtDevice, 0, seeds.size() * sizeof(o2::its::TrackITSExt), mGpuStreams[0].get()));
  checkGPUError(hipHostRegister(mTrackITSExt.data(), seeds.size() * sizeof(o2::its::TrackITSExt), hipHostRegisterPortable));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::downloadCellsDevice(const int layer)
{
  LOGP(debug, "gpu-transfer: downloading {} cells on layer: {}, for {} MB.", mCells[layer].size(), layer, mCells[layer].size() * sizeof(CellSeed) / MB);
  checkGPUError(hipMemcpyAsync(mCells[layer].data(), mCellsDevice[layer], mCells[layer].size() * sizeof(CellSeed), hipMemcpyDeviceToHost, mGpuStreams[0].get()));
  checkGPUError(hipHostUnregister(mCells[layer].data()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::downloadCellsNeighbours(std::vector<std::vector<std::pair<int, int>>>& neighbours, const int layer)
{
  LOGP(debug, "gpu-transfer: downloading {} neighbours, for {} MB.", neighbours[layer].size(), neighbours[layer].size() * sizeof(std::pair<int, int>) / MB);
  // TOOD: something less dangerous than assuming the same memory layout of std::pair and gpuPair... or not? :)
  checkGPUError(hipMemcpyAsync(neighbours[layer].data(), mNeighboursDevice[layer], neighbours[layer].size() * sizeof(gpuPair<int, int>), hipMemcpyDeviceToHost, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::downloadNeighboursLUT(std::vector<int>& lut, const int layer)
{
  LOGP(debug, "gpu-transfer: downloading {} neighbours lut, for {} MB.", lut.size(), lut.size() * sizeof(int) / MB);
  checkGPUError(hipMemcpyAsync(lut.data(), mNeighboursLUTDevice[layer], lut.size() * sizeof(int), hipMemcpyDeviceToHost, mGpuStreams[0].get()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::downloadTrackITSExtDevice(std::vector<CellSeed>& seeds)
{
  LOGP(debug, "gpu-transfer: downloading {} tracks, for {} MB.", mTrackITSExt.size(), mTrackITSExt.size() * sizeof(o2::its::TrackITSExt) / MB);
  checkGPUError(hipMemcpyAsync(mTrackITSExt.data(), mTrackITSExtDevice, seeds.size() * sizeof(o2::its::TrackITSExt), hipMemcpyDeviceToHost, mGpuStreams[0].get()));
  checkGPUError(hipHostUnregister(mTrackITSExt.data()));
  checkGPUError(hipHostUnregister(seeds.data()));
  // discardResult(hipDeviceSynchronize());
}

template <int nLayers>
void TimeFrameGPU<nLayers>::unregisterRest()
{
  LOGP(debug, "unregistering rest of the host memory...");
  checkGPUError(hipHostUnregister(mCells[0].data()));
  checkGPUError(hipHostUnregister(mCellsDevice.data()));
  checkGPUError(hipHostUnregister(mCellsLUTDevice.data()));
  for (auto iLayer{0}; iLayer < nLayers - 3; ++iLayer) {
    checkGPUError(hipHostUnregister(mCellsLookupTable[iLayer].data()));
  }
}
////////////////////////////////////////////////////////////////////////
/// Legacy
template <int nLayers>
void TimeFrameGPU<nLayers>::registerHostMemory(const int maxLayers)
{
  if (mHostRegistered) {
    return;
  } else {
    mHostRegistered = true;
  }
  for (auto iLayer{0}; iLayer < maxLayers; ++iLayer) {
    checkGPUError(hipHostRegister(mClusters[iLayer].data(), mClusters[iLayer].size() * sizeof(Cluster), hipHostRegisterPortable));
    checkGPUError(hipHostRegister(mNClustersPerROF[iLayer].data(), mNClustersPerROF[iLayer].size() * sizeof(int), hipHostRegisterPortable));
    checkGPUError(hipHostRegister(mIndexTables[iLayer].data(), (mStaticTrackingParams.ZBins * mStaticTrackingParams.PhiBins + 1) * mNrof * sizeof(int), hipHostRegisterPortable));
  }
  checkGPUError(hipHostRegister(mHostNTracklets.data(), (nLayers - 1) * mGpuParams.nTimeFrameChunks * sizeof(int), hipHostRegisterPortable));
  checkGPUError(hipHostRegister(mHostNCells.data(), (nLayers - 2) * mGpuParams.nTimeFrameChunks * sizeof(int), hipHostRegisterPortable));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::unregisterHostMemory(const int maxLayers)
{
  for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
    checkGPUError(hipHostUnregister(mTrackingFrameInfo[iLayer].data()));
  }
  checkGPUError(hipHostUnregister(mTrackingFrameInfoDevice.data()));
}

template <int nLayers>
void TimeFrameGPU<nLayers>::initialise(const int iteration,
                                       const TrackingParameters& trkParam,
                                       const int maxLayers,
                                       IndexTableUtils* utils,
                                       const TimeFrameGPUParameters* gpuParam)
{
  mGpuStreams.resize(mGpuParams.nTimeFrameChunks);
  o2::its::TimeFrame::initialise(iteration, trkParam, maxLayers);
}

template <int nLayers>
void TimeFrameGPU<nLayers>::wipe(const int maxLayers)
{
  unregisterHostMemory(maxLayers);
}

template <int nLayers>
void TimeFrameGPU<nLayers>::initDevice(IndexTableUtils* utils,
                                       const TrackingParameters& trkParam,
                                       const TimeFrameGPUParameters& gpuParam,
                                       const int maxLayers,
                                       const int iteration)
{
  // mStaticTrackingParams.ZBins = trkParam.ZBins;
  // mStaticTrackingParams.PhiBins = trkParam.PhiBins;
  // if (mFirstInit) {
  //   mGpuParams = gpuParam;
  //   allocMemAsync(reinterpret_cast<void**>(&mTrackingParamsDevice), sizeof(gpu::StaticTrackingParameters<nLayers>), nullptr, true);
  //   checkGPUError(hipMemcpy(mTrackingParamsDevice, &mStaticTrackingParams, sizeof(gpu::StaticTrackingParameters<nLayers>), hipMemcpyHostToDevice));
  //   if (utils) { // If utils is not nullptr, then its gpu vertexing
  //     mIndexTableUtils = *utils;
  //     allocMemAsync(reinterpret_cast<void**>(&mIndexTableUtilsDevice), sizeof(IndexTableUtils), nullptr, true);
  //   } else { // GPU tracking otherwise
  //     mIndexTableUtils.setTrackingParameters(trkParam);
  //   }

  // mMemChunks.resize(mGpuParams.nTimeFrameChunks, GpuTimeFrameChunk<nLayers>{static_cast<TimeFrame*>(this), mGpuParams});
  // mVerticesInChunks.resize(mGpuParams.nTimeFrameChunks);
  // mNVerticesInChunks.resize(mGpuParams.nTimeFrameChunks);
  // mLabelsInChunks.resize(mGpuParams.nTimeFrameChunks);
  // LOGP(info, "Size of fixed part is: {} MB", GpuTimeFrameChunk<nLayers>::computeFixedSizeBytes(mGpuParams) / MB);
  // LOGP(info, "Size of scaling part is: {} MB", GpuTimeFrameChunk<nLayers>::computeScalingSizeBytes(GpuTimeFrameChunk<nLayers>::computeRofPerChunk(mGpuParams, mAvailMemGB), mGpuParams) / MB);
  // LOGP(info, "Allocating {} chunks of {} rofs capacity each.", mGpuParams.nTimeFrameChunks, mGpuParams.nROFsPerChunk);

  // for (int iChunk{0}; iChunk < mMemChunks.size(); ++iChunk) {
  //   mMemChunks[iChunk].allocate(GpuTimeFrameChunk<nLayers>::computeRofPerChunk(mGpuParams, mGpuParams.maxGPUMemoryGB), mGpuStreams[iChunk]);
  // }
  //   for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
  //     allocMemAsync(reinterpret_cast<void**>(&mROframesClustersDevice[iLayer]), mROframesClusters[iLayer].size() * sizeof(int), nullptr, true);
  //     allocMemAsync(reinterpret_cast<void**>(&(mUsedClustersDevice[iLayer])), sizeof(unsigned char) * mGpuParams.clustersPerROfCapacity * mNrof, nullptr, true);
  //   }
  //   allocMemAsync(reinterpret_cast<void**>(&mVerticesDevice), sizeof(Vertex) * mGpuParams.maxVerticesCapacity, nullptr, true);
  //   allocMemAsync(reinterpret_cast<void**>(&mROframesPVDevice), sizeof(int) * (mNrof + 1), nullptr, true);

  //   mFirstInit = false;
  // }
  // if (maxLayers < nLayers) { // Vertexer
  //   for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
  //     checkGPUError(hipMemcpy(mROframesClustersDevice[iLayer], mROframesClusters[iLayer].data(), mROframesClusters[iLayer].size() * sizeof(int), hipMemcpyHostToDevice));
  //   }
  // } else { // Tracker
  //   checkGPUError(hipMemcpy(mVerticesDevice, mPrimaryVertices.data(), sizeof(Vertex) * mPrimaryVertices.size(), hipMemcpyHostToDevice));
  //   checkGPUError(hipMemcpy(mROframesPVDevice, mROframesPV.data(), sizeof(int) * mROframesPV.size(), hipMemcpyHostToDevice));
  //   if (!iteration) {
  //     for (auto iLayer{0}; iLayer < nLayers; ++iLayer) {
  //       checkGPUError(hipMemset(mUsedClustersDevice[iLayer], 0, sizeof(unsigned char) * mGpuParams.clustersPerROfCapacity * mNrof));
  //     }
  //   }
  // }
  // checkGPUError(hipMemcpy(mIndexTableUtilsDevice, &mIndexTableUtils, sizeof(IndexTableUtils), hipMemcpyHostToDevice));
}

template <int nLayers>
unsigned char* TimeFrameGPU<nLayers>::getDeviceUsedClusters(const int layer)
{
  return mUsedClustersDevice[layer];
}

template <int nLayers>
gsl::span<int> TimeFrameGPU<nLayers>::getHostNTracklets(const int chunkId)
{
  return gsl::span<int>(mHostNTracklets.data() + (nLayers - 1) * chunkId, nLayers - 1);
}

template <int nLayers>
gsl::span<int> TimeFrameGPU<nLayers>::getHostNCells(const int chunkId)
{
  return gsl::span<int>(mHostNCells.data() + (nLayers - 2) * chunkId, nLayers - 2);
}

template class TimeFrameGPU<7>;
} // namespace gpu
} // namespace its
} // namespace o2
